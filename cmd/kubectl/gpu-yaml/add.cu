#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <fstream>

using namespace std;

// m行N列
__global__ void add_gpu(int *c_matrixA, int *c_matrixB, int *c_matrixC, int n, int m) {
    int ix = threadIdx.x + blockDim.x*blockIdx.x;
	int iy = threadIdx.y + blockDim.y*blockIdx.y;
	unsigned int idx = iy * n + ix;
	if (ix < n && iy < m){
		c_matrixC[idx] = c_matrixA[idx] + c_matrixB[idx];
	}
}

vector<vector<int>> matrix_add(vector<vector<int>> &a, vector<vector<int>> &b) {
    const int m = a.size(), n = a[0].size();
    int *matrixA = (int *)malloc(sizeof(int) * m * n);
    int *matrixB = (int *)malloc(sizeof(int) * m * n);
    int *matrixC = (int *)malloc(sizeof(int) * m * n);

    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            matrixA[i * n + j] = a[i][j];
            matrixB[i * n + j] = b[i][j];
        }
    }

    int *c_matrixA, *c_matrixB, *c_matrixC;
    hipMalloc((void **)&c_matrixA, sizeof(int) * n * m);
    hipMalloc((void **)&c_matrixB, sizeof(int) * n * m);
    hipMalloc((void **)&c_matrixC, sizeof(int) * n * m);
    hipMemcpy(c_matrixA, matrixA, sizeof(int) * n * m, hipMemcpyHostToDevice);
    hipMemcpy(c_matrixB, matrixB, sizeof(int) * n * m, hipMemcpyHostToDevice);
    int dimx = 16;
    int dimy = 16;
	dim3 block(dimx, dimy);
    dim3 grid(n / block.x + 1, m / block.y + 1);
    add_gpu<<<grid, block>>>(c_matrixA, c_matrixB, c_matrixC, n, m);
    hipMemcpy(matrixC, c_matrixC, sizeof(int) * n * m, hipMemcpyDeviceToHost);
    vector<vector<int>> c(m, vector<int>(n,0));
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            c[i][j] = matrixC[i * n + j];
        }
    }
    free(matrixA);
    free(matrixB);
    free(matrixC);
    hipFree(c_matrixA);
    hipFree(c_matrixB);
    hipFree(c_matrixC);
    return c;
}

int main(int argc, char *argv[]) {
    vector<vector<int>> a{{1,2,3},{4,5,6},{7,8,9}}, b{{2,2,2},{2,2,2},{2,2,2}};
    vector<vector<int>> c = matrix_add(a, b);
    for (int i = 0; i < c.size(); i++){
        for (int j = 0; j < c[0].size(); j++){
            std::cout << c[i][j] << " ";
        }
        std::cout  << "\n";
    }

    return 0;
}
