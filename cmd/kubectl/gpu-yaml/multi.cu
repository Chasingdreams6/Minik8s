#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <fstream>

using namespace std;

__global__ void multi_gpu(int *d_matrixA, int *d_matrixB, int *d_matrixC, int x, int y)
{
    int ix = threadIdx.x + blockDim.x * blockIdx.x;
    int iy = threadIdx.y + blockDim.y * blockIdx.y;
    unsigned int idx = iy * x + ix;
    if (ix < x && iy < y)
    {
        int k;
        int sum = 0;
        for (k = 0; k < x; k++)
        {
            sum += d_matrixA[iy * x + k] * d_matrixB[k * x + ix];
        }
        d_matrixC[idx] = sum;
    }
}

vector<vector<int>> matrix_add(vector<vector<int>> &a, vector<vector<int>> &b)
{
    const int m = a.size(), n = a[0].size();
    int *matrixA = (int *)malloc(sizeof(int) * m * n);
    int *matrixB = (int *)malloc(sizeof(int) * m * n);
    int *matrixC = (int *)malloc(sizeof(int) * m * n);

    for (int i = 0; i < m; i++)
    {
        for (int j = 0; j < n; j++)
        {
            matrixA[i * n + j] = a[i][j];
            matrixB[i * n + j] = b[i][j];
        }
    }

    int *d_matrixA, *d_matrixB, *d_matrixC;
    hipMalloc((void **)&d_matrixA, sizeof(int) * n * m);
    hipMalloc((void **)&d_matrixB, sizeof(int) * n * m);
    hipMalloc((void **)&d_matrixC, sizeof(int) * n * m);
    hipMemcpy(d_matrixA, matrixA, sizeof(int) * n * m, hipMemcpyHostToDevice);
    hipMemcpy(d_matrixB, matrixB, sizeof(int) * n * m, hipMemcpyHostToDevice);
    int x = n, y = m;
    int dimx = 32;
    int dimy = 32;
    dim3 block(dimx, dimy);
    dim3 grid(x / block.x + 1, y / block.y + 1);
    multi_gpu<<<grid, block>>>(d_matrixA, d_matrixB, d_matrixC, x, y);
    hipMemcpy(matrixC, d_matrixC, sizeof(int) * n * m, hipMemcpyDeviceToHost);
    vector<int> temp(n, 0);
    vector<vector<int>> c(m, temp);
    for (int i = 0; i < m; i++)
    {
        for (int j = 0; j < n; j++)
        {
            c[i][j] = matrixC[i * n + j];
        }
    }
    free(matrixA);
    free(matrixB);
    free(matrixC);
    hipFree(d_matrixA);
    hipFree(d_matrixB);
    hipFree(d_matrixC);
    return c;
}

int main(int argc, char *argv[])
{
    vector<vector<int>> a{{1, 2, 3}, {4, 5, 6}, {7, 8, 9}}, b{{2, 2, 2}, {2, 2, 2}, {2, 2, 2}};
    vector<vector<int>> c = matrix_add(a, b);
    for (int i = 0; i < c.size(); i++)
    {
        for (int j = 0; j < c[0].size(); j++)
        {
            std::cout << c[i][j] << " ";
        }
        std::cout << "\n";
    }

    return 0;
}
